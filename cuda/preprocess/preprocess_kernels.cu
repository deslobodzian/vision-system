#include "hip/hip_runtime.h"
#include "preprocess_kernels.h"
#include "utils/logger.hpp"
// #include <opencv2/opencv.hpp>

static unsigned char *h_img = nullptr;
static unsigned char *d_bgr = nullptr;
static uchar3 *d_april_tag_bgr = nullptr;
static uchar3 *d_april_tag_decimated = nullptr;
static unsigned char *d_output = nullptr;

__global__ void kernel_convert_to_bgr(unsigned char *input,
                                      unsigned char *output, int width,
                                      int height, size_t stride) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  const int inIdx = y * stride + x * 4;
  const int outIdx = y * width * 3 + x * 3;

  if (inIdx + 3 >= stride * height || outIdx + 2 >= width * height * 3) {
    printf("Thread (%d, %d) out of bounds: inIdx = %d, outIdx = %d\n", x, y,
           inIdx, outIdx);
    return;
  }

  output[outIdx] = input[inIdx];
  output[outIdx + 1] = input[inIdx + 1];
  output[outIdx + 2] = input[inIdx + 2];
}

__global__ void kernel_preprocess_to_tensor(const unsigned char *d_bgr,
                                            float *d_output, int input_height,
                                            int input_width, int frame_s,
                                            int batch) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < input_width && y < input_height) {
    int i = y * input_width + x;

    uchar3 pixel =
        make_uchar3(d_bgr[i * 3], d_bgr[i * 3 + 1], d_bgr[i * 3 + 2]);

    d_output[batch * 3 * frame_s + i] = (float)pixel.z / 255.0;
    d_output[batch * 3 * frame_s + i + frame_s] = (float)pixel.y / 255.0;
    d_output[batch * 3 * frame_s + i + 2 * frame_s] = (float)pixel.x / 255.0;
  }
}

__global__ void kernel_convert_to_rgb(unsigned char *input,
                                      unsigned char *output, int width,
                                      int height, int step) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  const int inIdx = 4 * (y * step + x);    // BGRA input
  const int outIdx = 3 * (y * width + x);  // RGB output

  if (inIdx + 3 >= width * height * 4 || outIdx + 2 >= width * height * 3) {
    printf("Thread (%d, %d) out of bounds: inIdx = %d, outIdx = %d\n", x, y,
           inIdx, outIdx);
    return;
  }

  output[outIdx] = input[inIdx + 2];      // Red
  output[outIdx + 1] = input[inIdx + 1];  // Green
  output[outIdx + 2] = input[inIdx];      // Blue
}

__global__ void kernel_preprocess_letterbox(const unsigned char *d_bgr,
                                            unsigned char *d_output_image,
                                            int input_width, int input_height,
                                            int image_width, int image_height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  float r_w = (float)input_width / image_width;
  float r_h = (float)input_height / image_height;

  int w, h, x_offset, y_offset;
  if (r_h > r_w) {
    w = input_width;
    h = r_w * image_height;
    x_offset = 0;
    y_offset = (input_height - h) / 2;
  } else {
    w = r_h * image_width;
    h = input_height;
    x_offset = (input_width - w) / 2;
    y_offset = 0;
  }

  if (x < input_width && y < input_height) {
    uchar3 pixel;

    if (x >= x_offset && x < (x_offset + w) && y >= y_offset &&
        y < (y_offset + h)) {
      int x_resized = (int)((x - x_offset) * (image_width / (float)w));
      int y_resized = (int)((y - y_offset) * (image_height / (float)h));

      x_resized = min(x_resized, image_width - 1);
      y_resized = min(y_resized, image_height - 1);

      int i_resized = y_resized * image_width + x_resized;

      pixel = make_uchar3(d_bgr[i_resized * 3], d_bgr[i_resized * 3 + 1],
                          d_bgr[i_resized * 3 + 2]);
    } else {
      pixel.x = 128;
      pixel.y = 128;
      pixel.z = 128;
    }

    int i = y * input_width + x;
    d_output_image[i * 3] = pixel.x;
    d_output_image[i * 3 + 1] = pixel.y;
    d_output_image[i * 3 + 2] = pixel.z;
  }
}

void init_preprocess_resources(int image_width, int image_height,
                               int input_width, int input_height) {
  LOG_INFO("Allocating cuda memory");
  int max_image_width = image_width * 3;
  int max_image_height = image_height * 3;
  CUDA_CHECK(hipHostMalloc(
      &h_img, max_image_width * max_image_height * 3 * sizeof(unsigned char)));
  CUDA_CHECK(hipMalloc(
      &d_bgr, max_image_width * max_image_height * 3 * sizeof(unsigned char)));
  CUDA_CHECK(hipMalloc(
      &d_output, input_width * input_height * 3 * sizeof(unsigned char)));
}

void preprocess_sl(const sl::Mat &left_img, Tensor<float> &d_input,
                   hipStream_t &stream) {
  const int image_width = left_img.getWidth();
  const int image_height = left_img.getHeight();
  LOG_DEBUG(image_width, ", ", image_height);

  // BCWH
  const int batch = d_input.shape()[0] - 1;
  const int input_width = d_input.shape()[2];
  const int input_height = d_input.shape()[3];
  const size_t frame_s = input_width * input_height;
  LOG_DEBUG(d_input.print_shape());

  if (d_input.device() != Device::GPU) {
    d_input.to_gpu();
  }

  if (d_bgr == nullptr || d_output == nullptr) {
    init_preprocess_resources(image_width, image_height, input_width,
                              input_height);
  }
  hipError_t err;

  dim3 block(32, 32);
  dim3 grid_input((image_width + block.x - 1) / block.x,
                  (image_height + block.y - 1) / block.y);
  dim3 grid_output((input_width + block.x - 1) / block.x,
                   (input_height + block.y - 1) / block.y);

  kernel_convert_to_bgr<<<grid_input, block, 0, stream>>>(
      left_img.getPtr<sl::uchar1>(sl::MEM::GPU), d_bgr, image_width,
      image_height, left_img.getStepBytes(sl::MEM::GPU));
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel_convert_to_bgr launch failed: %s\n",
           hipGetErrorString(err));
    return;
  }

  kernel_preprocess_letterbox<<<grid_output, block, 0, stream>>>(
      d_bgr, d_output, input_width, input_height, image_width, image_height);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel_preprocess_and_letterbox launch failed: %s\n",
           hipGetErrorString(err));
    return;
  }

  kernel_preprocess_to_tensor<<<grid_output, block, 0, stream>>>(
      d_output, d_input.data(), input_height, input_width, frame_s, batch);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel_preprocess_to_tensor launch failed: %s\n",
           hipGetErrorString(err));
    return;
  }

  // unsigned char* h_letter = new unsigned char[input_width * input_height *
  // 3]; err = hipMemcpy(h_letter, d_output, input_width * input_height * 3 *
  // sizeof(unsigned char), hipMemcpyDeviceToHost); if (err != hipSuccess) {
  //     LOG_ERROR("CUDA memcpy Device to Host failed: ",
  //     hipGetErrorString(err)); return;
  // }

  // cv::Mat kernel_out(input_height, input_width, CV_8UC3, h_letter);
  // std::string filename = "kernel_letter_output.png";
  // cv::imwrite(filename, kernel_out);
  // delete[] h_letter;
}

void preprocess_cv(const cv::Mat &img, Tensor<float> &d_input,
                   hipStream_t &stream) {
  int image_width = img.cols;
  int image_height = img.rows;
  LOG_DEBUG(image_width, ", ", image_height);
  LOG_DEBUG(d_input.print_shape());

  if (d_input.device() != Device::GPU) {
    d_input.to_gpu();
  }
  // BCWH
  int batch = d_input.shape()[0] - 1;
  int input_width = d_input.shape()[2];
  int input_height = d_input.shape()[3];
  size_t frame_s = input_width * input_height;

  if (d_bgr == nullptr || d_output == nullptr || h_img == nullptr) {
    init_preprocess_resources(image_width, image_height, input_width,
                              input_height);
  }

  hipError_t err;

  size_t bytes = img.rows * img.cols * img.channels() * sizeof(unsigned char);

  // pinned memory so transfer is a bit faster
  memcpy(h_img, img.data, bytes);
  hipMemcpyAsync(d_bgr, h_img, bytes, hipMemcpyHostToDevice, stream);

  dim3 block(32, 32);
  dim3 grid_input((image_width + block.x - 1) / block.x,
                  (image_height + block.y - 1) / block.y);
  dim3 grid_output((input_width + block.x - 1) / block.x,
                   (input_height + block.y - 1) / block.y);

  kernel_preprocess_letterbox<<<grid_output, block, 0, stream>>>(
      d_bgr, d_output, input_width, input_height, image_width, image_height);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel_preprocess_and_letterbox launch failed: %s\n",
           hipGetErrorString(err));
    return;
  }

  kernel_preprocess_to_tensor<<<grid_output, block, 0, stream>>>(
      d_output, d_input.data(), input_height, input_width, frame_s, batch);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel_preprocess_to_tensor launch failed: %s\n",
           hipGetErrorString(err));
    return;
  }

  // unsigned char* h_letter = new unsigned char[input_width * input_height *
  // 3]; err = hipMemcpy(h_letter, d_output, input_width * input_height * 3 *
  // sizeof(unsigned char), hipMemcpyDeviceToHost); if (err != hipSuccess) {
  //     LOG_ERROR("CUDA memcpy Device to Host failed: ",
  //     hipGetErrorString(err)); return;
  // }

  // cv::Mat kernel_out(input_height, input_width, CV_8UC3, h_letter);
  // std::string filename = "kernel_letter_output.png";
  // cv::imwrite(filename, kernel_out);
  // delete[] h_letter;
}

// stupid I should just change to uchar3 for both but I'm lazy :(
__global__ void kernel_convert_to_bgr(unsigned char *input, uchar3 *output,
                                      int width, int height, size_t stride) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height) return;
  const int inIdx = y * stride + x * 4;
  const int outIdx = y * width + x;
  uchar3 bgrPixel;
  bgrPixel.x = input[inIdx];      // B
  bgrPixel.y = input[inIdx + 1];  // G
  bgrPixel.z = input[inIdx + 2];  // R
  output[outIdx] = bgrPixel;
}

__global__ void kernel_quad_decimate(uchar3 *input, uchar3 *output, int width,
                                     int height, int decimate) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int decimated_width = width / decimate;
  const int decimated_height = height / decimate;
  if (x >= decimated_width || y >= decimated_height) return;
  const int inIdx = y * decimate * width + x * decimate;
  const int outIdx = y * decimated_width + x;
  uchar3 decimated_pixel;
  decimated_pixel.x = (input[inIdx].x + input[inIdx + 1].x +
                       input[inIdx + width].x + input[inIdx + width + 1].x) /
                      4;
  decimated_pixel.y = (input[inIdx].y + input[inIdx + 1].y +
                       input[inIdx + width].y + input[inIdx + width + 1].y) /
                      4;
  decimated_pixel.z = (input[inIdx].z + input[inIdx + 1].z +
                       input[inIdx + width].z + input[inIdx + width + 1].z) /
                      4;
  output[outIdx] = decimated_pixel;
}

void init_april_tag_resources(int image_width, int image_height, int decimate) {
  LOG_INFO("Allocating cuda apriltag memory");
  CUDA_CHECK(hipMalloc(&d_april_tag_bgr,
                        image_width * image_height * sizeof(uchar3)));
  CUDA_CHECK(hipMalloc(
      &d_april_tag_decimated,
      (image_width / decimate) * (image_height / decimate) * sizeof(uchar3)));
}

void convert_sl_mat_to_april_tag_input(const sl::Mat &zed_mat,
                                       cuAprilTagsImageInput_t &tag_input,
                                       int decimate, hipStream_t &stream) {
  hipError_t err;
  if (zed_mat.getChannels() != 4 ||
      zed_mat.getDataType() != sl::MAT_TYPE::U8_C4) {
    LOG_ERROR("Unsupported sl::Mat format: Expected RGBA U8");
    return;
  }
  const int image_width = zed_mat.getWidth();
  const int image_height = zed_mat.getHeight();
  const size_t stride = zed_mat.getStepBytes(sl::MEM::GPU);
  if (d_april_tag_bgr == nullptr) {
    init_april_tag_resources(image_width, image_height, decimate);
  }
  dim3 block(16, 16);
  dim3 grid((image_width + block.x - 1) / block.x,
            (image_height + block.y - 1) / block.y);
  kernel_convert_to_bgr<<<grid, block, 0, stream>>>(
      zed_mat.getPtr<sl::uchar1>(sl::MEM::GPU), d_april_tag_bgr, image_width,
      image_height, stride);
  err = hipGetLastError();
  if (err != hipSuccess) {
    LOG_ERROR("kernel_convert_to_bgr launch failed: ", hipGetErrorString(err));
    return;
  }
  dim3 decimate_block(16, 16);
  dim3 decimate_grid(
      (image_width / decimate + decimate_block.x - 1) / decimate_block.x,
      (image_height / decimate + decimate_block.y - 1) / decimate_block.y);
  kernel_quad_decimate<<<decimate_grid, decimate_block, 0, stream>>>(
      d_april_tag_bgr, d_april_tag_decimated, image_width, image_height,
      decimate);
  err = hipGetLastError();
  if (err != hipSuccess) {
    LOG_ERROR("kernel_quad_decimate launch failed: ", hipGetErrorString(err));
    return;
  }
  tag_input.dev_ptr = d_april_tag_decimated;
  tag_input.pitch = 3 * (image_width / decimate);
  tag_input.width = static_cast<uint16_t>(image_width / decimate);
  tag_input.height = static_cast<uint16_t>(image_height / decimate);
}

void free_preprocess_resources() {
  CUDA_CHECK(hipHostFree(h_img));
  CUDA_CHECK(hipFree(d_bgr));
  CUDA_CHECK(hipFree(d_output));
}

void free_april_tag_resources() {
  CUDA_CHECK(hipFree(d_april_tag_bgr));
  CUDA_CHECK(hipFree(d_april_tag_decimated));
}
